#include "hip/hip_runtime.h"
﻿#include "CUDA_1.cuh"

using std::cout;
using std::endl;

MulCUDA::MulCUDA(void)
{

}

MulCUDA::~MulCUDA(void)
{

}

__global__ void helloWorld(char *str)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	str[idx] += idx;
}

char* MulCUDA::cudaExample(char *str)
{
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);	// one block per word
	dim3 dimBlock(6);	// one thread per character

	// invoke the kernel
	helloWorld << < dimGrid, dimBlock >> > (d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// free up the allocated memory on the device
	hipFree(d_str);

	return str;
}

__global__ void mulMatrixCUDA(float *P, float *M, float *N, int widthM, int widthN)
{
	int mBegin	= widthM * widthN * BLOCKSIZE;
	int mEnd	= mBegin + widthM - 1;
	int mStep	= BLOCKSIZE;
	
	int nBegin	= BLOCKSIZE * __cudaGet_blockIdx().x;
	int nStep	= BLOCKSIZE * widthN;

	float Csub	= 0;

	for (int mIndex = mBegin, nIndex = nBegin; mIndex <= mEnd; mIndex += mStep, nIndex += nStep)
	{
		__shared__ float Ms[BLOCKSIZE][BLOCKSIZE];
		__shared__ float Ns[BLOCKSIZE][BLOCKSIZE];

		Ms[__cudaGet_threadIdx().y][__cudaGet_threadIdx().x]
			= M[mBegin + widthM * __cudaGet_threadIdx().y + __cudaGet_threadIdx().x];

		Ns[__cudaGet_threadIdx().y][__cudaGet_threadIdx().x]
			= N[nBegin + widthN * __cudaGet_threadIdx().y + __cudaGet_threadIdx().x];


		for (int i = 0; i < BLOCKSIZE; ++i)
		{
			Csub += Ms[__cudaGet_threadIdx().y][i] * Ns[i][__cudaGet_threadIdx().x];
		}

	}


	int pIndex = widthM * BLOCKSIZE * __cudaGet_blockIdx().y + BLOCKSIZE * __cudaGet_blockIdx().x;
	P[pIndex + widthM * __cudaGet_threadIdx().y + __cudaGet_threadIdx().x];
}

