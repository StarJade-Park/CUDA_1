#include "hip/hip_runtime.h"
﻿#include "CUDA_1.cuh"

using std::cout;
using std::endl;

CMatrixMultiply::CMatrixMultiply(void)
{

}

CMatrixMultiply::~CMatrixMultiply(void)
{

}

__global__ void helloWorld(char *str)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	str[idx] += idx;
}

char* CMatrixMultiply::cuda_example(char *str)
{
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);	// one block per word
	dim3 dimBlock(6);	// one thread per character

	// invoke the kernel
	helloWorld <<< dimGrid, dimBlock >>> (d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// free up the allocated memory on the device
	delete[] d_str;
	hipFree(d_str);

	return str;
}

// Allocate device memory for M, N and P
// copy M and N to allocated device memory location
// Kernel invocation code to let the device perform the actual multiplication
// Read P from the device
// Free device matrices	​

bool CMatrixMultiply::MatrixMultiplyUsingCPU(const dim3& dimsM, const dim3& dimsN)‏
{
	// Allocate device memory for M, N and P
	// M
	int size_M = dimsM.x * dimsM.y * dimsM.z;		// BLOCKSIZE, BLOCKSIZE, 1
	int sizeOfMemory_M = size_M * sizeof(float);
	float* f_M = new float[sizeOfMemory_M];
	// N
	int size_N = dimsN.x * dimsN.y * dimsN.z;		// BLOCKSIZE, BLOCKSIZE, 1
	int sizeOfMemory_N = size_N * sizeof(float);
	float* f_N = new float[sizeOfMemory_N];
	// P
	dim3 dimsP;
	int sizeOfMemory_P = dimsP.x * dimsP.y * dimsP.z * sizeof(float);
	float* f_P = new float[sizeOfMemory_P];


	// copy M and N to allocated device memory location

	// Kernel invocation code to let the device perform the actual multiplication

	// Read P from the device

	// Free device matrices

}

bool CMatrixMultiply::MatrixMultiplyUsingCUDA(const dim3& dimsM, const dim3& dimsN)‏
{

}

bool CMatrixMultiply::matrixMultiplyUsingCUBLAS(const dim3& dimsM, const dim3& dimsN)
{
	return false;
}