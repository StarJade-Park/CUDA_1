#include "hip/hip_runtime.h"
#include "CUDA_1.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>
#include ""


GPUCUDA::GPUCUDA(void)
{

}

GPUCUDA::~GPUCUDA(void)
{
}

__global__ void helloWorld(char *str)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	str[idx] += idx;
}

char* GPUCUDA::cuda_example(char *str)
{
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);	// one block per word
	dim3 dimBlock(6);	// one thread per character

	// invoke the kernel
	helloWorld << < dimGrid, dimBlock >> > (d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// free up the allocated memory on the device
	hipFree(d_str);

	return str;
}