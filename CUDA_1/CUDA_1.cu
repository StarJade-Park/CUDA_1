#include "hip/hip_runtime.h"
﻿#include "CUDA_1.cuh"

using std::cout;
using std::endl;

CUDAExampleClass::CUDAExampleClass(void)
{

}

CUDAExampleClass::~CUDAExampleClass(void)
{

}

__global__ void helloWorld(char *str)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	str[idx] += idx;
}

char* CUDAExampleClass::cudaExample(char *str)
{
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);	// one block per word
	dim3 dimBlock(6);	// one thread per character

	// invoke the kernel
	helloWorld << < dimGrid, dimBlock >> > (d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// free up the allocated memory on the device
	hipFree(d_str);

	return str;
}