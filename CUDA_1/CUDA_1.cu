#include "hip/hip_runtime.h"
﻿#include "CUDA_1.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <hipfft/hipfft.h>
#include "hipblas.h"
#include <stdio.h>
#include <stdlib.h>
#include ""


GPUCUDA::GPUCUDA(void)
{

}

GPUCUDA::~GPUCUDA(void)
{
}

__global__ void helloWorld(char *str)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	str[idx] += idx;
}

__global__ void multiplicationKernel(float* lf, int Width)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	int idy = __cudaGet_blockIdx().y * __cudaGet_blockDim().y + __cudaGet_threadIdx().y;



}

char* GPUCUDA::cuda_example(char *str)
{
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);	// one block per word
	dim3 dimBlock(6);	// one thread per character

	// invoke the kernel
	helloWorld << < dimGrid, dimBlock >> > (d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// free up the allocated memory on the device
	hipFree(d_str);

	return str;
}

void GPUCUDA::MatrixMultiplication(float* M, float* N, float* P, int Width)‏
{
	int size = Width * Width * sizeof(float);
	float* Md, Nd, Pd;

	// Allocate device memory for M, N and P

	hipArray* cuArray;

	// copy M and N to allocated device memory location

	// Kernel invocation code to let the device perform the actual multiplication

	// Read P from the device

	// Free device matrices	​

}