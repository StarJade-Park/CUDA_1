#include "hip/hip_runtime.h"
﻿#include "CUDA_1.cuh"

using std::cout;
using std::endl;

CMatrixMultiply::CMatrixMultiply(void)
{

}

CMatrixMultiply::~CMatrixMultiply(void)
{

}

__global__ void helloWorld(char *str)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	str[idx] += idx;
}

char* CMatrixMultiply::cuda_example(char *str)
{
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);	// one block per word
	dim3 dimBlock(6);	// one thread per character

	// invoke the kernel
	helloWorld <<< dimGrid, dimBlock >>> (d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// free up the allocated memory on the device
	delete[] d_str;
	hipFree(d_str);

	return str;
}

// Allocate device memory for M, N and P
// copy M and N to allocated device memory location
// Kernel invocation code to let the device perform the actual multiplication
// Read P from the device
// Free device matrices	​

bool CMatrixMultiply::MatrixMultiplyUsingCPU(const dim3& dimsM, const dim3& dimsN)‏
{

}

bool CMatrixMultiply::MatrixMultiplyUsingCUDA(const dim3& dimsM, const dim3& dimsN)‏
{

}

bool CMatrixMultiply::matrixMultiplyUsingCUBLAS(const dim3& dimsM, const dim3& dimsN)
{
	return false;
}