#include "hip/hip_runtime.h"
﻿#include "CUDA_1.cuh"

using std::cout;
using std::endl;

CMatrixMultiply::CMatrixMultiply(void)
{

}

CMatrixMultiply::~CMatrixMultiply(void)
{

}

__global__ void helloWorld(char *str)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	str[idx] += idx;
}

__global__ void multiplicationKernel(float* lf, int Width)
{
	int idx = __cudaGet_blockIdx().x * __cudaGet_blockDim().x + __cudaGet_threadIdx().x;
	int idy = __cudaGet_blockIdx().y * __cudaGet_blockDim().y + __cudaGet_threadIdx().y;



}

char* CMatrixMultiply::cuda_example(char *str)
{
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	// set the grid and block sizes
	dim3 dimGrid(2);	// one block per word
	dim3 dimBlock(6);	// one thread per character

	// invoke the kernel
	helloWorld <<< dimGrid, dimBlock >>> (d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);

	// free up the allocated memory on the device
	delete[] d_str;
	hipFree(d_str);

	return str;
}


bool CMatrixMultiply::MatrixMultiplyUsingCPU()‏
{
	cout << "Using CPU " << endl;


}

bool CMatrixMultiply::MatrixMultiplyUsingCUDA()‏
{
	cout << "Using CUDA(GPU)" << endl;

	int size = Width * Width * sizeof(float);
	float* Md, Nd, Pd;

	// Allocate device memory for M, N and P
	hipArray* cuArray;

	// copy M and N to allocated device memory location

	// Kernel invocation code to let the device perform the actual multiplication

	// Read P from the device

	// Free device matrices	​

}
